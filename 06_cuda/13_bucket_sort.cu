#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>

__global__ void count_keys(int *keys, int *buckets, int n, int range) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    atomicAdd(&buckets[keys[index]], 1);
  }
}


__global__ void prefix_sum(int *buckets, int *sum, int range) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  for (int j=1; j<range; j<<=1) {
    sum[index] = buckets[index];
    __syncthreads();
    if (index >= j) {
      sum[index] += buckets[index-j];
    }
    __syncthreads();
  }
}

__global__ void fill_keys(int *keys, int *buckets, int *sum, int n, int range) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < range) {
    int start = sum[index];
    int key = index;
    for (int i = 0; i < buckets[index]; i++) {
      keys[start+i] = key;
    }
  }
}

int main() {
  int n = 50;
  int range = 5;
  int *keys, *buckets, *sum;
  int *d_keys, *d_buckets, *d_sum;

  keys = (int *)malloc(n * sizeof(int));
  buckets = (int *)calloc(range, sizeof(int));
  sum = (int *)malloc(range * sizeof(int));

  // d_keys, d_buckets are accessible from only GPU
  // cudaMalloc: allocate memory on GPU
  // cudaMallocManaged: allocate memory on unified memory
  hipMalloc(&d_keys, n * sizeof(int));
  hipMalloc(&d_buckets, range * sizeof(int));
  hipMalloc(&d_sum, range * sizeof(int));
  hipMemset(d_buckets, 0, range * sizeof(int)); // set 0 to all elements
  hipMemset(d_sum, 0, range * sizeof(int));     // set 0 to all elements

  for (int i = 0; i < n; i++) {
    keys[i] = rand() % range;
    printf("%d ", keys[i]);
  }
  printf("\n");

  // cudaMemcpy: copy memory from CPU to GPU
  hipMemcpy(d_keys, keys, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_buckets, buckets, range * sizeof(int), hipMemcpyHostToDevice);

  int blockSize = 1024;
  int numBlocks = (n + blockSize - 1) / blockSize;

  count_keys<<<numBlocks, blockSize>>>(d_keys, d_buckets, n, range);
  hipDeviceSynchronize();

  // cudaMemcpy: copy memory from GPU to CPU
  hipMemcpy(buckets, d_buckets, range * sizeof(int), hipMemcpyDeviceToHost);

  // scan
  // prefix_sum<<<numBlocks, blockSize>>>(d_buckets, d_sum, range);
  sum[0] = 0;
  for (int i = 1; i < range; i++) {
    sum[i] = sum[i-1] + buckets[i-1];
  }
  hipMemcpy(d_sum, sum, range * sizeof(int), hipMemcpyHostToDevice);

  fill_keys<<<numBlocks, blockSize>>>(d_keys, d_buckets, d_sum, n, range);
  hipDeviceSynchronize();

  // cudaMemcpy: copy memory from GPU to CPU
  hipMemcpy(keys, d_keys, n * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < n; i++) {
    printf("%d ", keys[i]);
  }
  printf("\n");

  free(keys);
  free(buckets);
  hipFree(d_keys);
  hipFree(d_buckets);

  return 0;
}
